#include "hip/hip_runtime.h"

#include <cuco/static_map.cuh>

#include <thrust/device_vector.h>
#include <thrust/equal.h>
#include <thrust/iterator/counting_iterator.h>
#include <thrust/iterator/transform_iterator.h>
#include <thrust/sequence.h>
#include <thrust/transform.h>

#include <hip/hip_cooperative_groups.h>

#include <cmath>
#include <cstddef>
#include <iostream>
#include <limits>
#include <fstream>
#include <sstream>

#include "data.hpp"
#include "timing.hpp"

struct __attribute__((packed)) tuple {
  int32_t start_index;
  int32_t length;

  constexpr __host__ __device__ tuple() : start_index(-1), length(-1) {}
  constexpr __host__ __device__ tuple(int32_t start, int32_t len) : start_index(start), length(len) {}
};

// Key which specifies the starting index and the length of a string
using Key   = tuple;
using Value = int;

// Function which takes a string where every line is a key and returns the keys
// based on the starting index and length of the string.
thrust::device_vector<Key> readFileToDeviceVector(const std::string& string) {
  std::istringstream stream(string);
  std::string line;

  thrust::device_vector<Key> keys;
  int32_t start_index = 0;

  while (std::getline(stream, line)) {
    if (line.empty()) continue;
    int32_t length = line.size();
    keys.push_back(Key(start_index, length));
    start_index += length + 1;
  }

  return keys;
}

// Function that reads a file and returns its contents as a char pointer.
std::string readFile(const char* filename) {
  std::ifstream file(filename, std::ios::binary);
  if (!file) {
    throw std::runtime_error("Failed to open file: " + std::string(filename));
  }
  std::ostringstream oss;
  oss << file.rdbuf();
  return oss.str();
}

struct hash_key {
  const char* string_keys;
  __host__ __device__ hash_key(const char* keys) : string_keys(keys) {}

  __host__ __device__ std::uint32_t operator()(Key const& key) const noexcept {
    int64_t start_index = key.start_index;
    int64_t length = key.length;
    std::uint32_t hash = 2166136261;

    for (int64_t i = 0; i < length; ++i) {
      char c = string_keys[start_index + i];
      hash = (hash * 16777619) ^ static_cast<std::uint32_t>(c);
    }
    return hash;
  }
};

struct equal_key {
  const char* string_keys;
  __host__ __device__ equal_key(const char* keys) : string_keys(keys) {}

  __device__ bool operator()(Key const& a, Key const& b) const {
    if (a.length != b.length) {
      return false;
    }
    
    bool result = true;
    for (int32_t i = 0; i < a.length; ++i) {
      result &= string_keys[a.start_index + i] == string_keys[b.start_index + i]; 
    }
    return result;
  }
};

template <typename MapRef, typename InputIterator, typename OutputIterator>
__global__ void scalar_find(MapRef set, InputIterator keys, std::size_t n, OutputIterator found) {
  int64_t i = blockDim.x * blockIdx.x + threadIdx.x;

  if (i < n) {
    auto [j, val] = *set.find(*(keys + i));
    found[i] = val;
  }
}

template <typename MapRef, typename InputIterator, typename OutputIterator>
__global__ void scalar_contains(MapRef set, InputIterator keys, std::size_t n, OutputIterator found) {
  int64_t i = blockDim.x * blockIdx.x + threadIdx.x;

  if (i < n) {
    bool b = set.contains(*(keys + i));
    found[i] = b;
  }
}

int main(int argc, char** argv) {
  if (argc != 3) {
    std::cerr << "Usage: " << argv[0] << " KEYS VALUES" << std::endl;
    return 1;
  }

  const char *keys_fname = argv[1];
  const char *vals_fname = argv[2];

  std::string string_keys = readFile(keys_fname);
  auto keys = readFileToDeviceVector(string_keys);
  auto keys_shape = keys.size();
  auto [vals, vals_shape] = read_futhark_array<int32_t,1>(vals_fname);

  char* d_string_keys;
  hipMalloc(&d_string_keys, string_keys.size() + 1);
  hipMemcpy(d_string_keys, string_keys.data(), string_keys.size() + 1, hipMemcpyHostToDevice);

  if (keys_shape != vals_shape[0]) {
    throw std::runtime_error("Mismatch in number of keys and values");
  }

  Key constexpr empty_key_sentinel     = tuple();
  Value constexpr empty_value_sentinel = -1;

  std::size_t num_keys = keys.size();

  std::cout << "n=" << num_keys << std::endl;

  auto constexpr load_factor = 0.5;
  std::size_t const capacity = std::ceil(num_keys / load_factor);

  auto map = cuco::static_map{
    capacity,
    cuco::empty_key{empty_key_sentinel},
    cuco::empty_value{empty_value_sentinel},
    equal_key{d_string_keys},
    cuco::linear_probing<1, hash_key>{hash_key{d_string_keys}}
  };
  
  thrust::device_vector<Key> insert_keys(keys.begin(), keys.end());
  thrust::device_vector<Value> insert_values(vals.begin(), vals.end());
  auto pairs = thrust::make_transform_iterator
  (thrust::counting_iterator<std::size_t>{0},
  cuda::proclaim_return_type<cuco::pair<Key, Value>>
  ([keys = insert_keys.begin(), values = insert_values.begin()] __device__(auto i) {
    return cuco::pair<Key, Value>{keys[i], values[i]};
  }));
  
  int insertAvgTime = measureAverageExecutionTime
  (2.0,
  [&]() {
    map.clear();
    map.insert(pairs, pairs + num_keys);
    hipDeviceSynchronize();
  });
  
  std::cout << "    construct: " << insertAvgTime << "μs" << std::endl;

    {
    thrust::device_vector<Value> found_values(num_keys);

    int lookupAvgTime = measureAverageExecutionTime
      (2.0,
       [&]() {
         map.find(insert_keys.begin(), insert_keys.end(), found_values.begin());
         hipDeviceSynchronize();
       });

    std::cout << "       lookup: " << lookupAvgTime << "μs" << std::endl;

    bool const all_values_match =
      thrust::equal(found_values.begin(), found_values.end(), insert_values.begin());

    if (!all_values_match) {
      std::cerr << "Did not find all values." << std::endl;
      return 1;
    }
  }

  {
    thrust::device_vector<Value> found_values(num_keys);

    const size_t BLOCK_SIZE = 256;
    size_t grid_size = (num_keys + BLOCK_SIZE - 1) / BLOCK_SIZE;

    int scalarLookupAvgTime = measureAverageExecutionTime
      (2.0,
       [&]() {
         scalar_find<<<grid_size,BLOCK_SIZE>>>(map.ref(cuco::find), insert_keys.begin(), num_keys, found_values.begin());
         hipDeviceSynchronize();
       });

    std::cout << "scalar lookup: " << scalarLookupAvgTime << "μs" << std::endl;

    bool const all_values_match =
      thrust::equal(found_values.begin(), found_values.end(), insert_values.begin());

    if (!all_values_match) {
      std::cerr << "Did not find all values." << std::endl;
      return 1;
    }
  }

  {
    thrust::device_vector<bool> exists(num_keys);

    int lookupAvgTime = measureAverageExecutionTime
      (2.0,
       [&]() {
         map.contains(insert_keys.begin(), insert_keys.end(), exists.begin());
         hipDeviceSynchronize();
       });

    std::cout << "       member: " << lookupAvgTime << "μs" << std::endl;

    bool const all_values_match =
      thrust::reduce(exists.begin(), exists.end(), true, thrust::logical_and<bool>());

    if (!all_values_match) {
      std::cerr << "Did not find all values." << std::endl;
      return 1;
    }
  }

  {
    thrust::device_vector<bool> exists(num_keys);

    const size_t BLOCK_SIZE = 256;
    size_t grid_size = (num_keys + BLOCK_SIZE - 1) / BLOCK_SIZE;

    int scalarLookupAvgTime = measureAverageExecutionTime
      (2.0,
       [&]() {
         scalar_contains<<<grid_size,BLOCK_SIZE>>>(map.ref(cuco::contains), insert_keys.begin(), num_keys, exists.begin());
         hipDeviceSynchronize();
       });

    std::cout << "scalar member: " << scalarLookupAvgTime << "μs" << std::endl;

    bool const all_values_match =
      thrust::reduce(exists.begin(), exists.end(), true, thrust::logical_and<bool>());

    if (!all_values_match) {
      std::cerr << "Did not find all values." << std::endl;
      return 1;
    }
  }

  hipFree(d_string_keys);
}
