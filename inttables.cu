#include "hip/hip_runtime.h"

#include <cuco/static_map.cuh>

#include <thrust/device_vector.h>
#include <thrust/equal.h>
#include <thrust/iterator/counting_iterator.h>
#include <thrust/iterator/transform_iterator.h>
#include <thrust/sequence.h>
#include <thrust/transform.h>

#include <hip/hip_cooperative_groups.h>

#include <cmath>
#include <cstddef>
#include <iostream>
#include <limits>

#include "data.hpp"
#include "timing.hpp"

using Key   = int64_t;
using Value = int;


template <typename MapRef, typename InputIterator, typename OutputIterator>
__global__ void scalar_find(MapRef set, InputIterator keys, std::size_t n, OutputIterator found) {
  int64_t i = blockDim.x * blockIdx.x + threadIdx.x;

  if (i < n) {
    auto [j, val] = *set.find(*(keys + i));
    found[i] = val;
  }
}

int main(int argc, char** argv) {
  if (argc != 3) {
    std::cerr << "Usage: " << argv[0] << " KEYS VALUES" << std::endl;
    return 1;
  }

  const char *keys_fname = argv[1];
  const char *vals_fname = argv[2];

  auto [keys, keys_shape] = read_futhark_array<int64_t,1>(keys_fname);
  auto [vals, vals_shape] = read_futhark_array<int32_t,1>(vals_fname);


  if (keys_shape[0] != vals_shape[0]) {
    throw std::runtime_error("Mismatch in number of keys and values");
  }

  Key constexpr empty_key_sentinel     = -1;
  Value constexpr empty_value_sentinel = -1;

  std::size_t num_keys = keys.size();

  std::cout << "n=" << num_keys << std::endl;

  auto constexpr load_factor = 0.5;
  std::size_t const capacity = std::ceil(num_keys / load_factor);

  auto map = cuco::static_map{
    capacity,
    cuco::empty_key{empty_key_sentinel},
    cuco::empty_value{empty_value_sentinel},
    cuda::std::equal_to<Key>(),
    cuco::linear_probing<1,cuco::default_hash_function<Key>>()
  };

  thrust::device_vector<Key> insert_keys(keys.begin(), keys.end());
  thrust::device_vector<Value> insert_values(vals.begin(), vals.end());
  auto pairs = thrust::make_transform_iterator
    (thrust::counting_iterator<std::size_t>{0},
     cuda::proclaim_return_type<cuco::pair<Key, Value>>
     ([keys = insert_keys.begin(), values = insert_values.begin()] __device__(auto i) {
       return cuco::pair<Key, Value>{keys[i], values[i]};
     }));

  int insertAvgTime = measureAverageExecutionTime
    (2.0,
     [&]() {
       map.clear();
       map.insert(pairs, pairs + num_keys);
       hipDeviceSynchronize();
     });

  std::cout << "    construct: " << insertAvgTime << "μs" << std::endl;

  {
    thrust::device_vector<Value> found_values(num_keys);

    int lookupAvgTime = measureAverageExecutionTime
      (2.0,
       [&]() {
         map.find(insert_keys.begin(), insert_keys.end(), found_values.begin());
         hipDeviceSynchronize();
       });

    std::cout << "       lookup: " << lookupAvgTime << "μs" << std::endl;

    bool const all_values_match =
      thrust::equal(found_values.begin(), found_values.end(), insert_values.begin());

    if (!all_values_match) {
      std::cerr << "Did not find all values." << std::endl;
      return 1;
    }
  }

  {
    thrust::device_vector<Value> found_values(num_keys);

    const size_t BLOCK_SIZE = 256;
    size_t grid_size = (num_keys + BLOCK_SIZE - 1) / BLOCK_SIZE;

    int scalarLookupAvgTime = measureAverageExecutionTime
      (2.0,
       [&]() {
         scalar_find<<<grid_size,BLOCK_SIZE>>>(map.ref(cuco::find), insert_keys.begin(), num_keys, found_values.begin());
         hipDeviceSynchronize();
       });

    std::cout << "scalar lookup: " << scalarLookupAvgTime << "μs" << std::endl;

    bool const all_values_match =
      thrust::equal(found_values.begin(), found_values.end(), insert_values.begin());

    if (!all_values_match) {
      std::cerr << "Did not find all values." << std::endl;
      return 1;
    }
  }
}
