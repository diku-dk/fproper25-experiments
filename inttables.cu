#include "hip/hip_runtime.h"

#include <cuco/static_map.cuh>

#include <thrust/device_vector.h>
#include <thrust/equal.h>
#include <thrust/iterator/counting_iterator.h>
#include <thrust/iterator/transform_iterator.h>
#include <thrust/sequence.h>
#include <thrust/transform.h>

#include <cmath>
#include <cstddef>
#include <iostream>
#include <limits>

#include "data.hpp"
#include "timing.hpp"

int main(int argc, char** argv) {
  using Key   = int64_t;
  using Value = int;

  if (argc != 3) {
    std::cerr << "Usage: " << argv[0] << " KEYS VALUES" << std::endl;
    return 1;
  }

  const char *keys_fname = argv[1];
  const char *vals_fname = argv[2];

  auto [keys, keys_shape] = read_futhark_array<int64_t,1>(keys_fname);
  auto [vals, vals_shape] = read_futhark_array<int32_t,1>(vals_fname);

  if (keys_shape[0] != vals_shape[0]) {
    throw std::runtime_error("Mismatch in number of keys and values");
  }

  Key constexpr empty_key_sentinel     = -1;
  Value constexpr empty_value_sentinel = -1;

  std::size_t num_keys = keys.size();

  std::cout << "n=" << num_keys << std::endl;

  auto constexpr load_factor = 0.5;
  std::size_t const capacity = std::ceil(num_keys / load_factor);

  auto map = cuco::static_map{
    capacity,
    cuco::empty_key{empty_key_sentinel},
    cuco::empty_value{empty_value_sentinel}
  };

  thrust::device_vector<Key> insert_keys(num_keys);
  thrust::sequence(keys.begin(), keys.end(), 0);
  thrust::device_vector<Value> insert_values(num_keys);
  thrust::sequence(vals.begin(), vals.end(), 0);
  auto pairs = thrust::make_transform_iterator
    (thrust::counting_iterator<std::size_t>{0},
     cuda::proclaim_return_type<cuco::pair<Key, Value>>
     ([keys = insert_keys.begin(), values = insert_values.begin()] __device__(auto i) {
       return cuco::pair<Key, Value>{keys[i], values[i]};
     }));


  int insertAvgTime = measureAverageExecutionTime
    (2.0,
     [&]() {
       map.clear();
       map.insert(pairs, pairs + num_keys);
       hipDeviceSynchronize();
     });

  std::cout << "    construct: " << insertAvgTime << "μs" << std::endl;

  thrust::device_vector<Value> found_values(num_keys);

  int lookupAvgTime = measureAverageExecutionTime
    (2.0,
     [&]() {
       map.find(insert_keys.begin(), insert_keys.end(), found_values.begin());
       hipDeviceSynchronize();
     });

  std::cout << "       lookup: " << lookupAvgTime << "μs" << std::endl;

  bool const all_values_match =
    thrust::equal(found_values.begin(), found_values.end(), insert_values.begin());

  if (!all_values_match) {
    return 1;
  }
}
